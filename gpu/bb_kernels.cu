#include "hip/hip_runtime.h"
#define PERBLOCK 4 // warps per block
#define TILE_SZ 32 // tile size

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime_api.h>
#include <assert.h>

// don't change order of following includes! (I should set up separate compilation...)
// GPU:common
#include "../include/gpu_helper.cuh"
#include "../include/reductions.cuh"
#include "../include/gpu_ivm_navi.cuh"

// GPU:bounds
#ifdef FSP
#include "gpu_fsp_bound.cuh"
#endif /* ifdef FSP */
#ifdef TEST
# include "gpu_test_bound.cuh"
#endif /* ifdef TEST */

#include "gpu_interval.cuh"
#include "loadbalance.cuh"

/*
 * countNodes_d : count decomposed nodes
 * counter_d : count current states
 */
// template <typename T>
__global__ void
goToNext_dense(int * jobMats_d, int * posVecs_d, int * endVecs_d, int * dirVecs_d, int * line_d, int * state_d, unsigned long long int * count, unsigned int * counter_d, int NN)
{
    int ivm = (blockIdx.x * blockDim.x + threadIdx.x);

    int state = state_d[ivm];
    int line  = line_d[ivm];

    int * pv   = &posVecs_d[ivm * size_d];
    int * ev   = &endVecs_d[ivm * size_d];
    int * jm   = jobMats_d + ivm * size_d * size_d; // global mem matrix
    int * mat_ptr = jm + line * size_d + *(pv + line);

    // initializing IVM
    if (state < 0) {
        if (*mat_ptr < 0) { // aka pruningCellState
            state = 1;
            for(int i=line+1;i<size_d;i++){
                pv[i] = 0;
            }
        }else{
            if (line < size_d - 2) {
                line++;
                generateLine2(jm, pv, dirVecs_d + ivm * size_d, line, state);
            } else {
                state = 1;
            }
        }
    }

    if (state > 0) {
        int l     = 0;          // first split [pos,end]
        while (pv[l] == ev[l] && l < size_d) l++;

        int * pos = pv + line; // current pos

        state = 0;
        // while (beforeEnd(pv, endv + warpID * size_d)) {
        while (pv[l] <= ev[l]) {              // approx check for (pos < end ?)
            //END OF LINE
            if (*pos >= (size_d - line)) {
                if (line == 0) break;      // cannot go up -> interval empty
                *pos = 0;
                line--;                                // aka goUp
                pos--;                                    // update current pos
                mat_ptr = jm + line * size_d + (*pos); // update pos in matrix (backtrack)
                *mat_ptr = negative_d(*mat_ptr);
            }
            else if (*mat_ptr < 0) // aka pruningCellState
            {
                assert(pv[line] < size_d);
                (*pos)++;  // aka goRight
                mat_ptr++; // update pos in matrix (next right)
            } else {
                assert(jm[line * size_d + pv[line]] >= 0);
                assert(line < size_d - 1);

                // found a node to bound --- check validity and set flag to "not empty"
                if (beforeEndPart(pv, ev, l)) {
                    atomicInc(&countNodes_d, INT_MAX);//atomic global counter
                    count[ivm]++;//per IVM counter
                    state = 1;
                }
                break;
            }
        }

        if (state == 1) {
            assert(line < size_d - 1);
            seqGoDown(jm, pv, &dirVecs_d[ivm*size_d], line);
            line++;
        }
    }

    // increment statistics counters
    if (state > 0) atomicInc(&counter_d[exploringState], INT_MAX);
    if (state == 0) atomicInc(&counter_d[emptyState], INT_MAX);
    if (state < 0) atomicInc(&counter_d[initState], nbIVM_d);

    state_d[ivm] = state;
    line_d[ivm]  = line;
}

/*
 * 4(N+2)*sizeof(int) shared memory
 *
 * countNodes_d : count decomposed nodes
 * counter_d : count current states
 */
template<unsigned NN>
__global__ void
goToNext2(int * jobMats_d, int * posVecs_d, int * endVecs_d, int * dirVecs_d, int * line_d, int * state_d, unsigned long long int * count, unsigned int * counter_d)
{
    thread_block_tile<32> g = tiled_partition<32>(this_thread_block());

    int ivm = (blockIdx.x * blockDim.x + threadIdx.x) / g.size(); // global ivm id
    int thPos = g.thread_rank();
    int warpID = threadIdx.x / g.size();

    //shared memory...... pos,end,state,line
    extern __shared__ int shar[];
    int * pv = shar;
    int * ev  = (int *) &pv[NN * size_d];
    int * state = (int *) &ev[NN * size_d];
    int * line  = (int *) &state[NN];

    // int * pv   = posv + warpID * size_d;            // shared mem - vector
    pv += warpID * size_d;
    ev += warpID * size_d;

    //load to shared memory ...
    for (int i = g.thread_rank(); i < size_d; i+=warpSize){
        pv[i] = posVecs_d[ivm * size_d + i];
        // posv[warpID * size_d + i] = posVecs_d[ivm * size_d + i];
        ev[i] = endVecs_d[ivm * size_d + i];
    }
    if (thPos == 0) {
        state[warpID] = state_d[ivm];
        line[warpID]  = line_d[ivm];
    }

    // pointers to IVM (just to compute less indices)
    g.sync();
    //=====================================
    int * jm   = jobMats_d + ivm * size_d * size_d; // global mem matrix

    // initializing IVM
    if (state[warpID] < 0)
        initStep(g,jm,pv,dirVecs_d + ivm * size_d,line[warpID],state[warpID]);
    g.sync();
    if (state[warpID] > 0) {
        exploreStep(g,jm,pv,ev,line[warpID],state[warpID]);
        g.sync();
        if (state[warpID] == 1) {
            count[ivm]++;//per IVM counter
            for (int i = g.thread_rank(); i< size_d; i += g.size()){
                assert(line[warpID] < size_d - 1);
                parallelGoDown(jm, pv, line[warpID], i, ivm);
            }
            g.sync();
            if (g.thread_rank() == 0){
                atomicInc(&countNodes_d, INT_MAX);//atomic global counter
                line[warpID]++;
            }
        }
    }
    g.sync();

    // increment statistics counters
    if (g.thread_rank() == 0) {
        if (state[warpID] > 0) atomicInc(&counter_d[exploringState], INT_MAX);
        if (state[warpID] == 0) atomicInc(&counter_d[emptyState], INT_MAX);
        if (state[warpID] < 0) atomicInc(&counter_d[initState], nbIVM_d);
    }

    g.sync();
//    __syncthreads();

    // back to global mem
    for (int i = thPos; i < size_d; i+=g.size()) {
        posVecs_d[ivm * size_d + i] = pv[i];
    }
    state_d[ivm] = state[warpID];
    line_d[ivm]  = line[warpID];
}

template<unsigned NN>
__global__ void
multistep_triggered(int * jobMats_d, int * posVecs_d, int * endVecs_d, int * dirVecs_d, int * line_d, int * state_d, unsigned long long int * count, unsigned int * counter_d, int *schedule_d, int* lim1_d, int*lim2_d, int*costsBE_d, int *flagLeaf, const int best,const int initialUB)
{
    thread_block_tile<32> g = tiled_partition<32>(this_thread_block());

    int ivm = (blockIdx.x * blockDim.x + threadIdx.x) / g.size(); // global ivm id
    int thPos = g.thread_rank();
    int warpID = threadIdx.x / g.size();

    //shared memory...... pos,end,state,line
    extern __shared__ int shar[];
    int *front = (int*)&shar;//partial schedule begin
    int *back    = (int *)&front[NN * _nbMachines];  // partial schedule end[M]
    int *remain  = (int *)&back[NN * _nbMachines];   // remaining work[M]
    // int * pv = shar;
    int * sched = (int *)&remain[NN * _nbMachines];
    // int * pv = (int *)&remain[NN * _nbMachines];
    int * pv = (int *)&sched[NN * size_d];
    int * ev  = (int *) &pv[NN * size_d];
    int * state = (int *) &ev[NN * size_d];
    int * line  = (int *) &state[NN];

    __shared__ int lim1[NN];
    __shared__ int lim2[NN];

    front += warpID*_nbMachines;
    back += warpID*_nbMachines;
    remain += warpID*_nbMachines;

    // int * pv   = posv + warpID * size_d;            // shared mem - vector
    sched += warpID * size_d;
    pv += warpID * size_d;
    ev += warpID * size_d;

    //load to shared memory ...
    for (int i = g.thread_rank(); i < size_d; i+=warpSize){
        pv[i] = posVecs_d[ivm * size_d + i];
        ev[i] = endVecs_d[ivm * size_d + i];
    }
    if (thPos == 0) {
        state[warpID] = state_d[ivm];
        line[warpID]  = line_d[ivm];
    }

    g.sync();
    //=====================================
    // (just to compute less indices)
    int * jm   = jobMats_d + ivm * size_d * size_d; // global mem matrix

    //fixed max number of steps...
    for(int i=0;i<500;i++){
        // initializing IVM
        if (state[warpID] < 0)
            initStep(g,jm,pv,dirVecs_d + ivm * size_d,line[warpID],state[warpID]);
        g.sync();
        if (state[warpID] > 0) {
            exploreStep(g,jm,pv,ev,line[warpID],state[warpID]);
            g.sync();
            if (state[warpID] == 1) {
                count[ivm]++;//per IVM counter
                for (int i = g.thread_rank(); i< size_d; i += g.size()){
                    assert(line[warpID] < size_d - 1);
                    assert(line[warpID] >= 0);
                    parallelGoDown(jm, pv, line[warpID], i, ivm);
                }
                g.sync();
                if (g.thread_rank() == 0)
                    line[warpID]++;
            }
        }
        g.sync();

        if(state[warpID]!=0){
            tile_decodeIVM(g, jm, pv, &dirVecs_d[ivm*size_d],line[warpID],lim1[warpID], lim2[warpID], sched);

            g.sync();
            tile_resetRemain(g, remain);
            tile_scheduleFront(g, sched, lim1[warpID], _tempsJob, front, remain);
            tile_scheduleBack(g, sched, lim2[warpID], _tempsJob, back, remain);
            g.sync();

            tile_addFrontAndBound(g,back,front,remain,&sched[lim1[warpID]+1],size_d-line[warpID],&costsBE_d[2 * ivm * size_d]);
            tile_addBackAndBound(g,back,front,remain,&sched[lim1[warpID]+1],size_d-line[warpID],&costsBE_d[(2 * ivm + 1) * size_d]);
            //
            int *jmrow = jm+line[warpID]*size_d;
            g.sync();

            if(g.thread_rank()==0){
                if (line[warpID] == size_d - 1) {
                    flagLeaf[ivm] = 1;
                    atomicInc(&targetNode, UINT_MAX);
                    jmrow[0] = negative_d(jmrow[0]);
                }
            }
            g.sync();

            if(flagLeaf[ivm]){
                if(g.thread_rank()==0)atomicInc(&_trigger,INT_MAX);
                break;
            }

            int dir=tile_MinBranch(g, jmrow, &costsBE_d[2 * ivm * size_d], &dirVecs_d[ivm*size_d], line[warpID],initialUB);
            dir=g.shfl(dir,0);
            g.sync();//!!!! every thread has dir

            if(thPos==0){
                if(line[warpID]==size_d-1){
                    jmrow[0] = negative_d(jmrow[0]);
                }
                //reverse
                int i1=0;
                int i2=size_d - line[warpID]-1;
                if(dirVecs_d[ivm*size_d+line[warpID]-1]!=dir){
                    while(i1<i2)
                    {
                        swap_d(&jmrow[i1],&jmrow[i2]);
                        i1++; i2--;
                    }
                    i1=lim1[warpID]+1;
                    i2=lim2[warpID]-1;
                    while(i1<i2)
                    {
                        swap_d(&sched[i1],&sched[i2]);
                        i1++; i2--;
                    }
                }
            }

            g.sync();
            tile_prune(g, jmrow, costsBE_d+2*ivm*size_d, dir, line[warpID], best);
        }else{
            if(g.thread_rank()==0)atomicInc(&_trigger,INT_MAX);
            break;
        }
        if(_trigger>2*nbIVM_d/10){
            break;
        }
    }

    // increment statistics counters
    if (g.thread_rank() == 0) {
        if (state[warpID] > 0) atomicInc(&counter_d[exploringState], INT_MAX);
        if (state[warpID] == 0) atomicInc(&counter_d[emptyState], INT_MAX);
        if (state[warpID] < 0) atomicInc(&counter_d[initState], nbIVM_d);
    }

    g.sync();
//    __syncthreads();

    // back to global mem
    for (int i = thPos; i < size_d; i+=g.size()) {
        posVecs_d[ivm * size_d + i] = pv[i];
    }
    state_d[ivm] = state[warpID];
    line_d[ivm]  = line[warpID];
}





template < typename T >
__global__ void
decodeIVMandFlagLeaf(const T *jobMats_d, const T *dirVecs_d, const T *posVecs_d, T *limit1s_d, T *limit2s_d, const T *line_d, T *schedules_d, T *state_d, int *todo_d, int *flagleaf)
{
    int ivm    = (blockIdx.x * blockDim.x + threadIdx.x) / warpSize; // global ivm id
    int warpID = threadIdx.x / warpSize;
    int thPos  = threadIdx.x % warpSize;                                           // threadIdx.x
                                                                     // % warpSize;
    extern __shared__ T decode_smem[];
    T *l1   = decode_smem;
    T *l2   = (T *)&l1[4];
    T *line = (T *)&l2[4];

    if (thPos == 0) {
        line[warpID] = line_d[ivm];
    }

    int *prmu = schedules_d + ivm * size_d;

    __syncthreads();

    int pointed, job;
    const int *jm = jobMats_d + ivm * size_d * size_d;

    // nothing to do
    if (state_d[ivm] == 0) return;

    //sequential
    if (thPos == 0) {
        l1[warpID] = -1;
        l2[warpID] = size_d;

        for (int j = 0; j < line[warpID]; j++) {
            pointed = posVecs_d[index2D(j, ivm)];
            job     = jm[j * size_d + pointed]; // jobMats_d[index3D(j, pointed,
                                                // ivm)];

            if (dirVecs_d[index2D(j, ivm)] == 0) {
                l1[warpID]++;
                prmu[l1[warpID]] = job;
            } else {
                l2[warpID]--;
                prmu[l2[warpID]] = job;
            }
        }
    }

    for(int l=thPos;l<size_d;l+=warpSize){
        schedules_d[index2D(l1[warpID] + 1 + l, ivm)] = jobMats_d[index3D(line_d[ivm], l, ivm)];
    }
    // for (int l = 0; l <= size_d / warpSize; l++) {
    //     if (l * warpSize + thPos < size_d - line_d[ivm]) {
    //         schedules_d[index2D(l1[warpID] + 1 + l * warpSize + thPos, ivm)] = jobMats_d[index3D(line_d[ivm], l * warpSize + thPos, ivm)];
    //     }
    // }

    if (thPos == 0) {
        if (line_d[ivm] == size_d - 1) {
            flagleaf[ivm] = 1;
            atomicInc(&targetNode, UINT_MAX);
        }
    }
    __threadfence();

    limit1s_d[ivm] = l1[warpID];
    limit2s_d[ivm] = l2[warpID];

    if (thPos == 0) {
        todo_d[ivm] = 0;

        if ((state_d[ivm] != 2) && (state_d[ivm] != 0)) {
            todo_d[ivm] = limit2s_d[ivm] - limit1s_d[ivm] - 1;
        }
    }
} // prepareSchedules

/*decode IVMs using one warp (thread_block_tile) per IVM

- decode operation is partially parallelized.
- resulting schedules in shared mem
*/
template < typename T >
__global__ void // __launch_bounds__(128, 16)
decodeIVM(const int *jobMats_d,const int *dirVecs_d,const int *posVecs_d,int *limit1s_d,int *limit2s_d,const T *line_d,int *schedules_d, const T *state_d)
{
    thread_block_tile<32> tile32 = tiled_partition<32>(this_thread_block());

    int ivm = (blockIdx.x * blockDim.x + threadIdx.x) / tile32.size(); // global ivm id
    int warpID = threadIdx.x / tile32.size();

    // SHARED MEMORY
    extern __shared__ int smemDecode[];
    int *prmu   = smemDecode;
    int *l1      = (int *)&prmu[4 * size_d];
    int *l2      = (int *)&l1[4];

    prmu += warpID * size_d;

    int line=line_d[ivm];
    const int *jm = jobMats_d + ivm * size_d * size_d;

    // nothing to do
    if (state_d[ivm] == 0) return;

    tile_decodeIVM(tile32, jm, &posVecs_d[ivm*size_d],&dirVecs_d[ivm*size_d],line, l1[warpID], l2[warpID], prmu);
    tile32.sync();

    //back to main mem
    for (int i = tile32.thread_rank(); i < size_d; i+=tile32.size()) {
        schedules_d[index2D(i,ivm)]=prmu[i];
    }
    limit1s_d[ivm] = l1[warpID];
    limit2s_d[ivm] = l2[warpID];
} // prepareSchedules

__global__ void
chooseBranchingSortAndPrune(int *jobMats_d,int *dirVecs_d,const int *posVecs_d,int *limit1s_d,int *limit2s_d, const int *line_d,int *schedules_d,int *costsBE_d,int *prio_d, int *state_d,int *todo_d,const int best,const int initialUB,const int branchStrategy)
{
    auto tile32 = tiled_partition<32>(this_thread_block());

    int ivm = (blockIdx.x * blockDim.x + threadIdx.x) / warpSize; // global ivm id
    int warpID = threadIdx.x / warpSize;
    int thPos = threadIdx.x % warpSize;

    // SHARED MEMORY
    extern __shared__ bool smemPrune[];
    int *jmrow = (int *)&smemPrune;   // schedule[N]
    int *prio      = (int *)&jmrow[4 * size_d];
    int *line    = (int *)&prio[4 * size_d];//&l2[4];

    jmrow += warpID * size_d;

    //load schedule limits and line to smem
    if (thPos == 0) {
        line[warpID] = line_d[ivm];
//        sum[warpID]  = 0;//999999;
    }
    tile32.sync();

    for (int i = tile32.thread_rank(); i < size_d; i+=warpSize) {
        jmrow[i]=jobMats_d[index3D(line[warpID],i,ivm)];
    }
    tile32.sync();

    // nothing to do
    if (state_d[ivm] == 0) return;

    int dir;
    switch(branchStrategy){
    case 1:{
        dir=tile_branchMaxSum<32>(tile32, jmrow, &costsBE_d[2 * ivm * size_d], &dirVecs_d[ivm*size_d], line[warpID]);
        break;}
    case 2:{
        dir=tile_MinBranch<32>(tile32, jmrow, &costsBE_d[2 * ivm * size_d], &dirVecs_d[ivm*size_d], line[warpID],initialUB);
        break;}
    case 3:{
        dir=tile_branchMinMin<32>(tile32, jmrow, &costsBE_d[2 * ivm * size_d], &dirVecs_d[ivm*size_d], line[warpID]);
        break;}
    }

    dir=tile32.shfl(dir,0);
    tile32.sync();//!!!! every thread has dir

    //order jobs in next row
    if(thPos==0){
        if(line[warpID]==size_d-1){
            jmrow[0] = negative_d(jmrow[0]);
        }

        //reverse
        int i1=0;
        int i2=size_d - line[warpID]-1;

        int prev_dir=(line[warpID]>0)?dirVecs_d[ivm*size_d+line[warpID]-1]:0;

        if(prev_dir!=dir){
            while(i1<i2)
            {
                swap_d(&jmrow[i1],&jmrow[i2]);
                i1++;
                i2--;
            }
        }
        if(prev_dir==1 && dir==0){
            for (int l = 0; l < size_d - line[warpID]; l++){
                schedules_d[ivm*size_d+limit1s_d[ivm]+1+l] = absolute_d(jmrow[l]);
            }
        }
    }

    tile32.sync();

    //prune
    tile_prune<32>(tile32, jmrow, costsBE_d+2*ivm*size_d, dir, line[warpID], best);

    //prapare strong bound
    if(_boundMode>=1){
        if (thPos == 0) {
            //popc ballot ... !
            todo_d[ivm] = 0;
            while (jmrow[todo_d[ivm]] >= 0 && todo_d[ivm] < size_d - line[warpID]) todo_d[ivm]++;//count non-pruned
        }
    }

    //back to main mem
    __syncthreads();
    for (int i = thPos; i < size_d; i+=warpSize) {
        jobMats_d[index3D(line[warpID],i,ivm)]=jmrow[i];
    }
}


template < typename T >
__global__ void // __launch_bounds__(128, 16)
ForwardBranchSortAndPrune(T *jobMats_d,T *dirVecs_d,const T *posVecs_d,T *limit1s_d,T *limit2s_d,
const T *line_d,T *schedules_d,int *costsBE_d,int *prio_d, T *state_d,int *todo_d,const int best,int *flagLeaf)
{
    thread_block_tile<32> tile32 = tiled_partition<32>(this_thread_block());

    int ivm = (blockIdx.x * blockDim.x + threadIdx.x) / warpSize; // global ivm id
    int warpID = threadIdx.x / warpSize;
    int thPos = threadIdx.x % warpSize;

    // SHARED MEMORY
    extern __shared__ bool smemPrune[];
    int *jmrow = (int *)&smemPrune;   // schedule[N]
    int *prio      = (int *)&jmrow[4 * size_d];
    int *line    = (int *)&prio[4 * size_d];//&l2[4];
//    int *sum     = (int *)&line[4];

    jmrow += warpID * size_d;
    int i;

    //load schedule limits and line to smem
    if (thPos == 0) {
        line[warpID] = line_d[ivm];
    }

    for (i = 0; i <= (size_d / warpSize); i++) {
        if (i * warpSize + thPos < size_d) {
            jmrow[i * warpSize + thPos]=jobMats_d[index3D(line[warpID],i * warpSize + thPos,ivm)];
        }
    }
    tile32.sync();

    // nothing to do
    if (state_d[ivm] == 0) return;

    int dir=0; //tile_chooseBranching(tile32, jmrow, &costsBE_d[2 * ivm * size_d], &dirVecs_d[ivm*size_d], line[warpID]);
    dir=tile32.shfl(dir,0);

    int job;
    for (i = 0; i <= (size_d / warpSize); i++) {
        if (i * warpSize + thPos < size_d - line[warpID]) {
            job = jmrow[i * warpSize + thPos];
            prio_d[ivm * size_d + job]=costsBE_d[(2 * ivm + (dir)) * size_d + job];
        }
    }

    if (thPos == 0) {
        // setting directionVector
        dirVecs_d[index2D(line[warpID], ivm)] = dir; // (p0>p1);

        //COMMENT OUT TO DISABLE SORTING
        // this is gnome sort (tested it: on small arrays better than insert,
        // quick, std::, bubble, selection sort !) O(n^2) worst case time, O(1)
        // gnomeSortSequential(jmrow,prio+warpID*size_d,1, size_d - line[warpID]);
    }

    if(thPos==0){
        if(line[warpID]==size_d-1){
            jmrow[0] = negative_d(jmrow[0]);
        }
    }

    tile_prune(tile32, jmrow, costsBE_d+2*ivm*size_d, dir, line[warpID], best);

    if (thPos == 0) {
        todo_d[ivm] = 0;
        while (jmrow[todo_d[ivm]] >= 0 && todo_d[ivm] < size_d - line[warpID]) todo_d[ivm]++;//count non-pruned
    }

    __syncthreads();
    for (i = 0; i <= (size_d / warpSize); i++) {
        if (i * warpSize + thPos < size_d) {
            jobMats_d[index3D(line[warpID],i * warpSize + thPos,ivm)]=jmrow[i * warpSize + thPos];
            //=jobMats_d[index3D(line[warpID],i * warpSize + thPos,ivm)];
        }
    }
}



template < typename T >
__global__ void
prune2noSort(T *jobMats_d, const T *dirVecs_d, const T *line_d,
       const int *costsBE_d, const T *state_d, const int best)
{
    // thread indexing
    int ivm   = (blockIdx.x * blockDim.x + threadIdx.x) / warpSize; // global
                                                                    // ivm id
    int thPos = threadIdx.x % warpSize;

    int dir = dirVecs_d[index2D(line_d[ivm], ivm)];
    int l   = 0;

    // pruning
    if (state_d[ivm] != 0) {
        for (l = 0; l <= size_d / warpSize; l++) {
            if (l * warpSize + thPos < size_d - line_d[ivm]) {
                int job = jobMats_d[index3D(line_d[ivm], l * warpSize + thPos, ivm)];

                if (job < 0) continue;                            // already pruned in phase I

                int val = costsBE_d[index2D(job, 2 * ivm + dir)]; // LB
#ifdef FINDALL
                if (val > best) {                                 // find ALL optimal solutions
#else  /* ifdef FINDALL */
                if (val >= best) {
#endif /* ifdef FINDALL */
                    jobMats_d[index3D(line_d[ivm], l * warpSize + thPos, ivm)] = negative_d(job); // eliminate node !
                }
            }
        }// for warp
    } // if(state)
}

//

/***********************************************************************/
template < typename T >
__global__ void prune(T *jobMats_d, T *dirVecs_d, const T *line_d,
                      const int *costsBE_d, const  int *sums_d, const T *state_d, unsigned int *ctrl_d, unsigned int *counter_d, int best) {
    /**** thread indexing ***********/
    int ivm   = (blockIdx.x * blockDim.x + threadIdx.x) / warpSize; // global
                                                                    // ivm id
    int thPos = threadIdx.x % warpSize;                             // threadId
                                                                    // within
                                                                    // IVM

    //  int best = (int)ctrl_d[4];

    // setting directionVector
    if (thPos == 0) {
        if (sums_d[2 * ivm] > sums_d[2 * ivm + 1]) {
            dirVecs_d[index2D(line_d[ivm], ivm)] = 0;
        } else {
            dirVecs_d[index2D(line_d[ivm], ivm)] = 1;
        }
    }

    __syncthreads();

    // pruning
    int l = 0;

    if (state_d[ivm] != 0) {
        for (l = 0; l <= size_d / warpSize; l++) {
            if (l * warpSize + thPos < size_d - line_d[ivm]) {
                int job = absolute_d(jobMats_d[index3D(line_d[ivm], l * warpSize + thPos, ivm)]);
                int val = (1 - dirVecs_d[index2D(line_d[ivm], ivm)]) *
                          costsBE_d[index2D(job, 2 * ivm)] +
                          dirVecs_d[index2D(line_d[ivm], ivm)] *
                          costsBE_d[index2D(job, 2 * ivm + 1)];

                if (val >= best) {
                    jobMats_d[index3D(line_d[ivm], l * warpSize + thPos, ivm)] = negative_d(
                        jobMats_d[index3D(line_d[ivm], l * warpSize + thPos, ivm)]); //
                                                                                     // eliminate
                                                                                     // node
                                                                                     // !
                }
            }
        }// for warp
    }

    if (threadIdx.x == 0) {
        counter_d[exploringState] = 0;
        counter_d[emptyState]     = 0;
        counter_d[initState]      = 0;
    }
}

template < typename T >
__global__ void sortedPrune(T *jobMats_d, T *dirVecs_d, const T *line_d,
                            const int *costsBE_d, const  int *sums_d, const T *state_d, int *flagLeaf, const int best) {
    /**** thread indexing ***********/
    int ivm   = (blockIdx.x * blockDim.x + threadIdx.x) / warpSize; // global
                                                                    // ivm id
    int thPos = threadIdx.x % warpSize;                             // threadId
                                                                    // within
                                                                    // IVM
    int dir;

    // setting directionVector
    if (sums_d[2 * ivm] > sums_d[2 * ivm + 1]) {
        dir                                  = 0;
        dirVecs_d[index2D(line_d[ivm], ivm)] = 0;
    } else {
        dir                                  = 1;
        dirVecs_d[index2D(line_d[ivm], ivm)] = 1;
    }

    __syncthreads();

    // SORTING (insert sort)
    // int l=0;
    if (state_d[ivm] != 0) {
        // for(l=0; l<=size_d/warpSize; l++){
        if (thPos == 0) {
            int i, j, key;

            for (i = 1; i < size_d - line_d[ivm]; i++) {       // for(i=1;i<N;i++)
                key = jobMats_d[index3D(line_d[ivm], i, ivm)]; // arr[i];
                j   = i;

                // while(j>0 && arr[j-1] > key)
                while (j > 0 && costsBE_d[index2D(jobMats_d[index3D(line_d[ivm], j - 1, ivm)], 2 * ivm + dir)] > costsBE_d[index2D(key, 2 * ivm + dir)]) {
                    jobMats_d[index3D(line_d[ivm], j, ivm)] = jobMats_d[index3D(line_d[ivm], j - 1, ivm)]; //
                                                                                                           // arr[j]=arr[j-1];
                    j--;
                }
                jobMats_d[index3D(line_d[ivm], j, ivm)] = key;
            }
        }
    }

    __syncthreads();

    int l = 0;

    // int jobs[MAXJOBS];

    //  int job = absolute(jobMats_d[index3D(line_d[ivm], l*warpSize + thPos,
    // ivm)]);
    // __syncthreads();

    // pruning
    if (state_d[ivm] != 0) {
        for (l = 0; l <= size_d / warpSize; l++) {
            if (l * warpSize + thPos < size_d - line_d[ivm]) {
                //        jobs[thPos] = absolute(jobMats_d[index3D(line_d[ivm],
                // l*warpSize + thPos, ivm)]);
                int job = absolute_d(jobMats_d[index3D(line_d[ivm], l * warpSize + thPos, ivm)]);

                // int val = costsBE_d[index2D(jobs[thPos], 2 * ivm + dir)];
                int val = costsBE_d[index2D(job, 2 * ivm + dir)];
#ifdef FINDALL
                if (val > best) {
#else
                if (val >= best) {
#endif

                    //  jobMats_d[index3D(line_d[ivm], l*warpSize + thPos, ivm)]
                    // = negative(jobs[thPos]);
                    jobMats_d[index3D(line_d[ivm], l * warpSize + thPos, ivm)] = negative_d(job); //
                                                                                                  // eliminate
                                                                                                  // node
                                                                                                  // !
                }
            }
        }// for warp
    }
}

// ------------------------------------------------------------------------------------------------------------------------------------------------------------------------
template < typename T >
__global__ void prepareBound(T* schedule_d, int* costsBE_d, T* dirVecs_d,T* line_d,T *limit1s_d, T *limit2s_d, int *todo_d, int *ivmId_d, int *toSwap_d, int *tempArr_d)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int ivm   = tid;// / warpSize;

    int job,dir;

//    if(thPos==0){
        dir=dirVecs_d[index2D(line_d[ivm],ivm)];
        for(int i=limit1s_d[ivm] + 1;i<limit2s_d[ivm];i++){
            job=schedule_d[index2D(i,ivm)];
            if(costsBE_d[index2D(job,2*ivm+dir)]>=0){
                ivmId_d[tempArr_d[ivm] + i]  = ivm;
                toSwap_d[tempArr_d[ivm] + i] = i;//limit1s_d[ivm] + 1 + i;
            }
        }

    if (tid == 0) {
        todo = tempArr_d[nbIVM_d - 1] + todo_d[nbIVM_d - 1];
    }
}


template < typename T >
__global__ void
prepareBound2(T *limit1s_d, T *limit2s_d, int *todo_d, int *ivmId_d, int *toSwap_d, int *tempArr_d)
{
    int thPos = threadIdx.x % warpSize;
    int tid   = blockIdx.x * blockDim.x + threadIdx.x;

    // int ivm = blockIdx.x * PERBLOCK + threadIdx.x / warpSize;
    int ivm = tid / warpSize;

    int l = 0;

    for (l = 0; l <= size_d / warpSize; l++) {
        if (l * warpSize + thPos < todo_d[ivm]) {
            ivmId_d[tempArr_d[ivm] + l * warpSize + thPos]  = ivm; // ivm;
            toSwap_d[tempArr_d[ivm] + l * warpSize + thPos] = limit1s_d[ivm] + 1 + l * warpSize + thPos;
        }
    }

    if (tid == 0) {
        todo = tempArr_d[nbIVM_d - 1] + todo_d[nbIVM_d - 1];
    }
}

template < typename T >
__global__ void
share_inter_gpu2(int den, const T *mat, const T *pos, T *end,
                 const T *dir, const T *line, const T *state, T *steal_mat, T *steal_pos, T *steal_end,
                 T *steal_dir, T *steal_line,
                 T *steal_state, int toDivide)
{
    int thPos = threadIdx.x % warpSize;
    int ivm   = (blockIdx.x * blockDim.x + threadIdx.x) / warpSize;

    int i = 0;
    int l = 0;

    if (ivm < toDivide) {
        // IF thief empty && victim exploring && victim interval large enough...
        if ((steal_state[ivm] == 0) && (state[ivm] == 1)) {
            // ... THEN share
            while (pos[index2D(l, ivm)] == end[index2D(l, ivm)] && l < line[ivm] && l < size_d - 3) l++;

            if (pos[index2D(l, ivm)] >= end[index2D(l, ivm)]) {
                for (int k = 0; k <= size_d / warpSize; k++) {
                    if (k * warpSize + thPos < size_d) {
                        steal_pos[index2D(k * warpSize + thPos, ivm)] = size_d - (k * warpSize + thPos) - 1;
                        steal_end[index2D(k * warpSize + thPos, ivm)] = size_d - (k * warpSize + thPos) - 1;
                    }
                }

                if (thPos == 0) {
                    steal_pos[index2D(thPos, ivm)] = size_d;
                    steal_state[ivm]               = 0;
                }
                steal_state[ivm] = 0;
            } else {
                for (int k = 0; k <= size_d / warpSize; k++) {
                    if (k * warpSize + thPos < l) {
                        steal_pos[index2D(k * warpSize + thPos, ivm)] = pos[index2D(k * warpSize + thPos, ivm)];
                        steal_dir[index2D(k * warpSize + thPos, ivm)] = dir[index2D(k * warpSize + thPos, ivm)];
                    }

                    for (i = 0; i < l; i++) {
                        if (k * warpSize + thPos <
                            size_d) steal_mat[index3D(i, k * warpSize + thPos,
                                                      ivm)] = mat[index3D(i, k * warpSize + thPos, ivm)];
                    }

                    if (k * warpSize + thPos < size_d) {
                        steal_end[index2D(k * warpSize + thPos, ivm)]    = end[index2D(k * warpSize + thPos, ivm)];
                        steal_mat[index3D(l, k * warpSize + thPos, ivm)] = mat[index3D(l, k * warpSize + thPos, ivm)];
                    }
                }

                if (thPos == 0) {
                    steal_dir[index2D(l, ivm)] = dir[index2D(l, ivm)];
                    steal_pos[index2D(l, ivm)] = cuttingPosition(l, den, pos + ivm * size_d, end + ivm * size_d,
                                                                 mat + ivm * size_d * size_d);
                    end[index2D(l, ivm)] = steal_pos[index2D(l, ivm)] - 1;

                    for (i = l + 1; i < size_d; i++) {
                        steal_pos[index2D(i, ivm)] = 0;
                        end[index2D(i, ivm)]       = size_d - i - 1;
                    }
                    steal_line[ivm]  = l;
                    steal_state[ivm] = 1;
                }
            }
        }
    }
} // share_inter_gpu2

template < typename T >
__global__ void setRoot(T *mat, T *dir) {
    int thPos = threadIdx.x % warpSize;
    int ivm   = (blockIdx.x * blockDim.x + threadIdx.x) / warpSize;

    for (int l = thPos; l < size_d; l+=warpSize) {
        mat[index3D(0, l, ivm)] = root_d[l];
    }

    if (thPos == 0) dir[index2D(0, ivm)] = root_dir_d;
}

//
//
//
//          FLOWSHOP
//
//
#ifdef FSP
template < typename T >
__global__ void __launch_bounds__(128, 8) boundJohnson(const T * schedules_d, const T * limit1s_d, const T * limit2s_d, const T * line_d, int * costsBE_d, int * sums_d, const T * state_d, const int * toSwap_d, const int * ivmId_d, unsigned int * bdleaves_d, unsigned int * ctrl_d, int * flagLeaf, const int best) {
    /**** thread indexing ****/
    register int tid   = blockIdx.x * blockDim.x + threadIdx.x;
    register int BE    = tid & 1;
    register int ivmnb = ivmId_d[(tid >> 1)]; // the ivm tid is working on

    /***** shared memory declarations *****/
    extern __shared__ unsigned char sharedArray[];
    unsigned char *tempsJob_sh = (unsigned char *)sharedArray;
    char *permut_sh            = (char *)&tempsJob_sh[_nbMachines * size_d];

    if (threadIdx.x < size_d) {
        for (int j = 0; j < _nbMachines; j++) tempsJob_sh[j * size_d + threadIdx.x] =
                (unsigned char)_tempsJob[j * size_d + threadIdx.x];
    }

    //  if (tid < 2 * ctrl_d[toDo]) {
    if (tid < 2 * todo) {
        if (tid % 2 == 0) {
            for (int i = 0; i < size_d; i++) permut_sh[index2D(i, threadIdx.x >> 1)] =
                    schedules_d[index2D(i, ivmnb)];
        }
    }

    __syncthreads();

    /*******************************************/

    //  if (tid < 2 * ctrl_d[toDo]) {
    if (tid < 2 * todo) {
        char limit1 = limit1s_d[ivmnb] + 1 - BE;
        char limit2 = limit2s_d[ivmnb] - BE;

        char Swap1 = toSwap_d[(tid >> 1)];
        char Swap2 = (1 - BE) * limit1 + BE * limit2;

        char jobnb = permut_sh[index2D(Swap1, threadIdx.x >> 1)];

        int where = ivmnb * 2 * size_d + BE * size_d + (int)jobnb;

        if (line_d[ivmnb] < (size_d - 1)) { // boundNodes
            costsBE_d[where] = computeCost(permut_sh, Swap1, Swap2, limit1, limit2, _tempsJob, threadIdx.x >> 1, _tabJohnson, best);

            //      costsBE_d[where] = computeCost(permut_sh, Swap1, Swap2,
            // limit1, limit2, tempsJob_sh, threadIdx.x >> 1, _tabJohnson);// +
            // BE * limit2;
            atomicAdd(&sums_d[2 * ivmnb + BE], costsBE_d[where]);
        } else if (BE == 0) { // boundLeaves
            //      char pos = posVecs_d[index2D(line_d[ivmnb], ivmnb)];
            //      jobMats_d[index3D(line_d[ivmnb], pos, ivmnb)] =
            //          negative(jobMats_d[index3D(line_d[ivmnb], pos, ivmnb)]);
            if (state_d[ivmnb] == 1) bdleaves_d[ivmnb]++;

            flagLeaf[ivmnb] = 1;
            atomicInc(&ctrl_d[foundLeaf], UINT_MAX);
        }
    }
}
#endif /* ifdef FSP */
